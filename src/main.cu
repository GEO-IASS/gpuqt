#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------------80
    This is the main function of GPUQT (GPU Quantum Transport).

    Authors: Zheyong Fan <brucenju@gmail.com> <zheyongfan@163.com> 
             Ville Vierimaa <ville.v.vierimaa@aalto.fi>
------------------------------------------------------------------------------*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>

#include "common.h"
#include "gpuqt.h"

//----------------------------------------------------------------------------80
int main(int argc, char *argv[])
{	
    if (argc != 2)
    {
        std::cout << "Usage: src/gpuqt input.txt" << std::endl;
        exit(1);
    }
	
    std::ifstream input(argv[1]);
    if (!input.is_open())
    {
        std::cout << "Failed to open " << argv[1] << std::endl;
        exit(1);
    }		

    std::string directory;
    while (std::getline(input, directory))
    {
 		if (directory == "")
 			continue;
        std::cout << std::endl;
        std::cout << "===========================================" << std::endl;
        std::cout << "Run KGQT simulation for " << directory << std::endl; 
        std::cout << "===========================================" << std::endl;

        clock_t time_begin = clock();
         
        // call the driver function
        gpuqt(directory);

        clock_t time_finish = clock();
        real time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;

        std::cout << std::endl;
        std::cout << "===========================================" << std::endl;
        std::cout << "Total time used for " << directory << " = " << time_used <<" s" << std::endl; 
        std::cout << "===========================================" << std::endl;
    }

    return 0;
}

