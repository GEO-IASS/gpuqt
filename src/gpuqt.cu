#include "hip/hip_runtime.h"
#include "gpuqt.h"
#include "vector.h"
#include "hamiltonian.h"
#include "sigma.h"
#include "model.h"


void gpuqt(std::string input_directory)
{
    // Initialize model on the CPU
    Model model(input_directory);
    Hamiltonian H(model);
    Vector random_state(model);

    clock_t time_begin, time_finish;
    real time_used;
    
    // Loop over different random vectors
    for (int i = 0; i < model.number_of_random_vectors; ++i)
    {
        std::cout << "Starting vector number " << i+1 << std::endl;
        model.initialize_state(random_state);

        // Always calculate the DOS, since it is very cheap
        time_begin = clock(); 
        find_dos(model, H, random_state);
        time_finish = clock();
        time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
        std::cout << "Time used for finding DOS = " << time_used << " s" << std::endl; 

        // Calculate the MSD only if you want to
        if (model.calculate_msd == 1)  
        {    
            time_begin = clock();
            find_msd(model, H, random_state);
            time_finish = clock();
            time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
    	    std::cout << "Time used for finding MSD = " << time_used << " s" << std::endl;
        }

        // Calculate the VAC only if you want to
        if (model.calculate_vac == 1)  
        {
            time_begin = clock();
            find_vac(model, H, random_state);
            time_finish = clock();
            time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
	        std::cout << "Time used for finding VAC = " << time_used << " s" << std::endl;
        }
    }  
      	
}


